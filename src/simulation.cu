#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <SFML/Graphics.hpp>
#include "particle.h"
#include <algorithm>
#include <hip/hip_math_constants.h>

using namespace std;
__global__ void resetDensity(Particle* particles, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    particles[index].density = 0;
}

//__________________________________________________________________________________________________________________________________________________________________________________
// Kernel to compute interactions of every particle to every other particle using parallel computation
__global__ void collisionKernel(Particle* particles, int size)
{
    int threadNum = blockIdx.x * blockDim.x + threadIdx.x;                                      // Set the index of own particle and other particle
    int i = threadNum / size;                                                                   
    int j = threadNum % size;

    if (i >= size || j >= size || i == j) return;                                               // Checking if out of range or self

    float dx = particles[i].pos.x - particles[j].pos.x;                                         // Diff in x
    float dy = particles[i].pos.y - particles[j].pos.y;                                         // Diff in y

    float r2 = dx * dx + dy * dy;                                                         //
    float h = 20.0f;                                                                            // Radius of effect
    float h2 = h * h;                                                                           //

    if (r2 < h2) {
        float diff = h2 - r2;
        float poly6Coeff = 4.0f / (HIP_PI_F * powf(h, 8.0f));
        float influence = poly6Coeff * powf(h2,3);

        atomicAdd(&particles[i].density, influence);
    }
}

//__________________________________________________________________________________________________________________________________________________________________________________
// Kernel to compute local physics like wall collisions or gravity to save on resources
__global__ void generalKenel(Particle* particles, int size, int threadPerBlock, int screenSize, int mouseX, int mouseY)
{
    float gravity = 0;
    int index = blockIdx.x * blockDim.x + threadIdx.x;                                          // Compute index (directly correlated to particle in particles)
    if(index<size){
        // Keep particles in bounds
        if (particles[index].pos.x <= 0){
            particles[index].pos.x = 0;
            particles[index].vel.x *= -.9;
        }else if (particles[index].pos.x >= screenSize)
        {
            particles[index].pos.x = screenSize;
            particles[index].vel.x *= -.9;
        }else if (particles[index].pos.y <= 0)
        {
            particles[index].pos.y = 0;
            particles[index].vel.y *= -.9;
        }else if (particles[index].pos.y >= screenSize-1)
        {
            particles[index].pos.y = screenSize-1;
            particles[index].vel.y *= -.9;
        }
        // Apply Gravity
        particles[index].vel.y -= gravity;
        //update position based on velocity
        particles[index].pos.x += particles[index].vel.x;   
        particles[index].pos.y += particles[index].vel.y; 
    }  
}

//__________________________________________________________________________________________________________________________________________________________________________________

void compute(vector<Particle>& particles, int screenSize, int totalParticles, sf::Vector2i mousePos){

    int size = particles.size();


    Particle* dParticles;                                                                   // Create empty pointer that will hold pointer to gpu mem
    hipMalloc(&dParticles, size*sizeof(Particle));                                         // Allocate space in gpu mem and store pointer to it in dParticles
    hipMemcpy(dParticles,particles.data(),size*sizeof(Particle),hipMemcpyHostToDevice);   // Copy over data into space allocated at the space dParticles points to 
    

    // Compute how many threads/blocks will be needed
    int threadPerBlock = 1024;
    int numBlocksCollision = totalParticles*totalParticles/threadPerBlock+1;
    int numBlocksGen = totalParticles/threadPerBlock+1;


    // Send to gpu to compute
    resetDensity<<<numBlocksGen, threadPerBlock>>>(dParticles, size);
    hipDeviceSynchronize();

    collisionKernel<<<numBlocksCollision, threadPerBlock>>>(dParticles, size);
    hipDeviceSynchronize();

    generalKenel<<<numBlocksGen, threadPerBlock>>>(dParticles, size,threadPerBlock, screenSize, mousePos.x, mousePos.y);
    hipDeviceSynchronize();                                                                // Wait for GPU to finish before accessing results
    

    hipMemcpy(particles.data(),dParticles,size*sizeof(Particle),hipMemcpyDeviceToHost);
    hipFree(dParticles);

    for (size_t i = 0; i < particles.size(); i++)
    {
        float normalizedDensity = min(particles[i].density*4000.0f, 255.0f);
        unsigned char colorValue = static_cast<unsigned char>(normalizedDensity);

        // Set color: Red base, green intensity changes with density
        particles[i].circle.color.r = 255;
        particles[i].circle.color.g = colorValue;
        particles[i].circle.color.b = 0;
    }
}

//__________________________________________________________________________________________________________________________________________________________________________________

vector<Particle> createParticles(int totalParticles, int screenSize){
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> posDistrib(0, screenSize); // random positions
    uniform_real_distribution<float> velDistrib(-1.f, 1.f);   // random velocities

    vector<Particle> particles;
    particles.reserve(totalParticles);

    for (int i = 0; i < totalParticles; i++) {

        float xpos = posDistrib(gen);
        float ypos = posDistrib(gen);
        float xvel = velDistrib(gen);
        float yvel = velDistrib(gen);
        particles.emplace_back(Particle({xpos, ypos}, {xvel, yvel}));
    }

    return particles;
}

// vector<Particle> createParticles(int totalParticles, int screenSize) {
//     const int spacing = 7; // space between particles in pixels

//     // Estimate grid size (try to make it as square as possible)
//     int cols = static_cast<int>(sqrt(totalParticles));
//     int rows = (totalParticles + cols - 1) / cols; // round up

//     vector<Particle> particles;
//     particles.reserve(totalParticles);

//     // Center the grid on screen
//     int gridWidth = cols * spacing;
//     int gridHeight = rows * spacing;
//     int startX = (screenSize - gridWidth) / 2;
//     int startY = (screenSize - gridHeight) / 2;

//     for (int i = 0; i < totalParticles; ++i) {
//         int row = i / cols;
//         int col = i % cols;

//         float xpos = startX + col * spacing;
//         float ypos = startY + row * spacing;

//         float xvel = 0;
//         float yvel = 0;

//         particles.emplace_back(Particle({xpos, ypos}, {xvel, yvel}));
//     }

//     return particles;
// }
//__________________________________________________________________________________________________________________________________________________________________________________